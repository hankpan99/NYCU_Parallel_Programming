#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(int *device_data, float lowerX, float lowerY, float stepX, float stepY, size_t pitch, int maxIterations){
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    
    // process index
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    // initialize mandel variables
    float c_re = lowerX + thisX * stepX;
    float c_im = lowerY + thisY * stepY;
    float z_re = c_re, z_im = c_im;

    // pointer points to the pixel should be processed in this thread
    int* ptr = (int*) ((char*) device_data + thisY * pitch) + thisX;

    // by theorem in mandel, if |c| <= 0.25 then c belongs to M
    if(z_re * z_re + z_im * z_im <= 0.25f){
        *ptr = maxIterations;
        return;
    }
    
    // mandel iteration
    int intensity;
    for(intensity = 0; intensity < maxIterations; intensity++){
        if(z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }
    
    *ptr = intensity;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations){
    // compute steps
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    // allocate memory
    int N = resX * resY;
    int *host_data;
    hipHostAlloc((void**) &host_data, N * sizeof(int), hipHostMallocMapped);

    int *device_data;
    size_t pitch;
    hipMallocPitch(&device_data, &pitch, resX * sizeof(int), resY);

    // launch kernel function
    dim3 threads_per_block(20, 20);
    dim3 num_blocks(resX / threads_per_block.x, resY / threads_per_block.y);
    mandelKernel<<<num_blocks, threads_per_block>>>(device_data, lowerX, lowerY, stepX, stepY, pitch, maxIterations);
    
    // output answers
    hipMemcpy2D(host_data, resX * sizeof(int), device_data, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy(img, host_data, N * sizeof(int));
    
    // free memory
    hipFree(device_data);
    hipHostFree(host_data);
}